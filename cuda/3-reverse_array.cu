#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <stdlib.h>
#define N 2048
#define THREAD_PER_BLOCK 512

__global__ void reverseArray (int *d_b , int *d_a)
{
  int old_id = threadIdx.x + blockIdx.x * blockDim.x; 
  int new_id = N - 1 - old_id ; 
  d_b[old_id] = d_a[new_id];
}

int main (){
   int *h_a, *d_a, *d_b;
   int size = N * sizeof (int); 
   h_a = (int *) malloc (size);

   for (int i = 0; i < N; i++)
   {
       h_a[i] = i;
   }

   hipMalloc ((void **) &d_a, size); 
   hipMalloc ((void **) &d_b, size); 
   hipMemcpy (d_a, h_a, size, hipMemcpyHostToDevice);
 
   reverseArray <<< N/THREAD_PER_BLOCK, THREAD_PER_BLOCK >>>(d_b, d_a);

   hipMemcpy (h_a, d_b, size, hipMemcpyDeviceToHost);

   hipFree(d_a); hipFree(d_b); 

   for (int i = 0; i < N; i++)
   {
       printf("%d\n", h_a[i]);
   }


   free(h_a);

   return 0;}