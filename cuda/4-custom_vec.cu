#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <stdlib.h>
#define BLOCKSIZE 16
#define SIZE 128

__global__ void vectvectshared (int *A, int *B, int *r)
{    
  __shared__ int temp[SIZE];
	
  int i = threadIdx.x; 
  int j = threadIdx.y;

  int ind  = j + (blockDim.x * i);

  if (ind < SIZE)
     temp[ind] = A[ind] * B[ind];
    
   __syncthreads();

   if(ind == 0){
     int sum = 0;
     for(int i = 0; i < SIZE; i++)
        sum += temp[i];
     *r = sum;
   }
}

void fill_dp_vector (int* vec,int size)
{
   int ind;
   for(ind = 0; ind < size; ind++)
        vec[ind] = 3*ind;
}

int main ()
{
   int *hostA, *hostB, *res;
   int *devA, *devB, *devres;

   int vlen;

   vlen=SIZE;
	
   dim3 threadspblock(BLOCKSIZE,BLOCKSIZE);

   hostA = (int *) malloc (vlen * sizeof(int));
   hostB = (int *) malloc (vlen * sizeof(int));
   res = (int *) malloc (sizeof(int));
   
   fill_dp_vector (hostA, vlen);   
   fill_dp_vector (hostB, vlen);

   hipMalloc((void **) &devA, vlen * sizeof(int));
   hipMalloc((void **) &devB, vlen * sizeof(int));
   hipMalloc((void **) &devres, sizeof(int));

   hipMemcpy(devA, hostA, vlen * sizeof(int), hipMemcpyHostToDevice);
   hipMemcpy(devB, hostB, vlen * sizeof(int), hipMemcpyHostToDevice);
		
   vectvectshared<<<1, threadspblock>>>(devA, devB, devres);

   hipMemcpy (res, devres, sizeof(int), hipMemcpyDeviceToHost);
 
   hipFree (devA);
   hipFree (devB);
   hipFree (devres);

   printf("%d\n", *res);

   free (hostA);
   free (hostB);
   free (res);

   return 0;}