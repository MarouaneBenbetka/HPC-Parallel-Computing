#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <stdlib.h>
#define N 512

__global__ void dot (int *a , int *b , int *c) 
{
   __shared__ int temp[N]; 	                                                             
   temp[threadIdx.x] = a[threadIdx.x] * b[threadIdx.x]; 
   
   __syncthreads ();

   // Le thread 0 effectue la somme 
   if (threadIdx.x == 0) {
      int sum = 0;
	 for (int i = 0; i < N; i++)
           sum += temp[i];
      *c = sum;        
   }} 

int main (){
   int *a, * b, *c;
   int *gpu_a, *gpu_b, *gpu_c;
   int size = N * sizeof (int); 
      
   hipMalloc ((void **) &gpu_a, size); 
   hipMalloc ((void **) &gpu_b, size);
   hipMalloc ((void **) &gpu_c, sizeof (int)); 
   
   a = (int *) malloc (size);
   b = (int *) malloc (size);
   c = (int *) malloc (sizeof (int)); 

   for (int i = 0; i < N; i++)
      {
       a[i] = i;
      }


   for (int i = 0; i < N; i++)
      {
       b[i] = 2*i;
      }

   hipMemcpy (gpu_a, a, size, hipMemcpyHostToDevice); 
   hipMemcpy (gpu_b, b, size, hipMemcpyHostToDevice);
 
   dot <<<1, N>>> (gpu_a, gpu_b, gpu_c);

   hipMemcpy (c, gpu_c, sizeof(int), hipMemcpyDeviceToHost);

   hipFree(gpu_a); hipFree(gpu_b); hipFree(gpu_c);

   printf("%d\n", *c);

   free(a); free(b); free(c);

   return 0;
}