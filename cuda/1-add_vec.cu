#include "hip/hip_runtime.h"
%% cu
#include <stdio.h>
#include <stdlib.h>
#define N 1000
#define THREAD_PER_BLOCK 512

__global__ void add(int *a, int *b, int *c)
{
    int indice = threadIdx.x + blockIdx.x * blockDim.x;
    if (indice < N)
        c[indice] = a[indice] + b[indice];
}

int main()
{
    int *a, *b, *c;
    int *gpu_a, *gpu_b, *gpu_c;
    int size = N * sizeof(int);

    hipMalloc((void **)&gpu_a, size);
    hipMalloc((void **)&gpu_b, size);
    hipMalloc((void **)&gpu_c, size);

    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    for (int i = 0; i < N; i++)
    {
        a[i] = i;
    }

    for (int i = 0; i < N; i++)
    {
        b[i] = 2 * i;
    }

    hipMemcpy(gpu_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(gpu_b, b, size, hipMemcpyHostToDevice);

    add<<<(N + THREAD_PER_BLOCK) / THREAD_PER_BLOCK, THREAD_PER_BLOCK>>>(gpu_a, gpu_b, gpu_c);

    hipMemcpy(c, gpu_c, size, hipMemcpyDeviceToHost);

    hipFree(gpu_a);
    hipFree(gpu_b);
    hipFree(gpu_c);

    for (int i = 0; i < N; i++)
    {
        printf("%d\n", c[i]);
    }

    free(a);
    free(b);
    free(c);

    return 0;
}